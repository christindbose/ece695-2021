#define TILE_WIDTH 32

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width)
{
  __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
  __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;

  int Row = by * blockDim.y + ty;
  int Col = bx * blockDim.x + tx;
  float Pvalue = 0;

  // Loop over the M and N tiles required to compute the P element
  for (int p = 0; p < Width/TILE_WIDTH; ++p) {
    // Collaborative loading of M and N tiles into shared memory
    ds_M[ty][tx] = M[Row*Width + p*TILE_WIDTH+tx];
    ds_N[ty][tx] = N[(p*TILE_WIDTH+ty)*Width + Col];
    __syncthreads();

    for (int i = 0; i < TILE_WIDTH; ++i)
      Pvalue += ds_M[ty][i] * ds_N[i][tx];
    __syncthreads();
  }
  P[Row*Width+Col] = Pvalue;
}


#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
  const int size = 1024;
  float* M;
  float* N;
  float* P;
  hipMallocManaged(&M, size*size*sizeof(float));
  hipMallocManaged(&N, size*size*sizeof(float));
  hipMallocManaged(&P, size*size*sizeof(float));

  MatrixMulKernel<<<(size/TILE_WIDTH, size/TILE_WIDTH),
                    (TILE_WIDTH, TILE_WIDTH)>>>(M,N,P,size);
  hipDeviceSynchronize();
  std::cout << P[0];

}

